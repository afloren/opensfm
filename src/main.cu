#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <conio.h>
#include "Timer.h"

__global__ void ProjectPoints(float* P, float3* X, float2* xh)
{
   int i = threadIdx.x;

   __shared__ float Ps[12];

   if(i<12)
   {
	   Ps[i] = P[i];
   }

   __syncthreads();

   //float x = Ps[0]*X[3*i]+Ps[1]*X[3*i+1]+Ps[2]*X[3*i+2]+Ps[3];
   //float y = Ps[4]*X[3*i]+Ps[5]*X[3*i+1]+Ps[6]*X[3*i+2]+Ps[7];
   //float z = Ps[8]*X[3*i]+Ps[9]*X[3*i+1]+Ps[10]*X[3*i+2]+Ps[11];
   //xh[2*i] = x/z;
   //xh[2*i+1] = y/z;

   float x = Ps[0]*X[i].x + Ps[1]*X[i].y + Ps[2]*X[i].z + Ps[3];
   float y = Ps[4]*X[i].x + Ps[5]*X[i].y + Ps[6]*X[i].z + Ps[7];
   float z = Ps[8]*X[i].x + Ps[9]*X[i].y + Ps[10]*X[i].z + Ps[11];

   xh[i].x = x/z;
   xh[i].y = y/z;
}

__global__ void ReprojectionError(float* P, float* X, float* x, float *r)
{
   int i = threadIdx.x;
   float xhx = P[0]*X[3*i]+P[1]*X[3*i+1]+P[2]*X[3*i+2]+P[3];
   float xhy = P[4]*X[3*i]+P[5]*X[3*i+1]+P[6]*X[3*i+2]+P[7];
   float xhz = P[8]*X[3*i]+P[9]*X[3*i+1]+P[10]*X[3*i+2]+P[11];
   r[2*i] = x[2*i]-xhx/xhz;
   r[2*i+1] = x[2*i+1]-xhy/xhz;
}

void printVector(float* A, int size)
{
   std::cout << "[ ";
   for(int i=0;i<size;i++)
   {
      std::cout << A[i] << " ";
   }


   std::cout << "]" << std::endl;
}

void randomVector(float* A, int size)
{
   for(int i=0;i<size;i++)
   {
      A[i] = rand()/(float)RAND_MAX;
   }
}

int main()
{
   Timer timer;
   const int N = 512;
   float P[12] = {1,0,0,0,0,1,0,0,0,0,1,0};
   float Q[N*3];
   float q[N*2];
    
   //printVector(P,12);

   randomVector(Q,N*3);
   //printVector(Q,N*3);

   float* dP;
   hipMalloc(&dP,sizeof(float)*12);
   float3* dQ;
   hipMalloc(&dQ,sizeof(float3)*N);
   float2* dq;
   hipMalloc(&dq,sizeof(float2)*N);

   timer.start();
   
   hipMemcpy(dP,P,sizeof(float)*12,hipMemcpyHostToDevice);  
   hipMemcpy(dQ,Q,sizeof(float3)*N,hipMemcpyHostToDevice);

   std::cout << "Elapsed time(s): " << timer.getElapsed() << std::endl;

   ProjectPoints<<<1,N>>>(dP,dQ,dq);

   std::cout << "Elapsed time(s): " << timer.getElapsed() << std::endl;

   hipMemcpy(q,dq,sizeof(float2)*N,hipMemcpyDeviceToHost);

   std::cout << "Elapsed time(s): " << timer.getElapsed() << std::endl;

   //printVector(q,N*2);

   getch();
}


